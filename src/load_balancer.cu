#include "load_balancer.h"

#include <hip/hip_runtime.h>


void LoadBalancer::getDeviceCount() {
	hipGetDeviceCount(&this->gpuCount);
}

void LoadBalancer::synchronize() {
	hipDeviceSynchronize();
}
