#include "hip/hip_runtime.h"
#include <cstdio>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cstdint>
#include "MaxReduction.cu"
#include "Sort.cu"
#include "BinarySearch.cu"
#include <iostream>
#include "PassManager.hpp"
#include <omp.h>

template<class Callable, class... Args>
void timeWrapper(Callable f, Args... args) {
	auto start = std::chrono::steady_clock::now();
	f(args...);
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	printf("%f\n", elapsed_seconds.count());
}

int main() {
	const int config_number = 5;
	const int config_start = 0;
	const int gpu = 0;
	int config = config_start;

	const int power_of_two_number = 10;
	const int power_of_two_start = 0;
	int power_of_two = power_of_two_start;

	const int multiplier_of_512_number = 8;
	const int multiplier_of_512_start = 2;
	int multiplier_of_512 = multiplier_of_512_start;

	const int small_test_number = 10;
	const int big_test_number = 4;
	int limit = big_test_number;


	printf("SORT - FLOAT\n");
	for (config = config_start; config < config_number; config++)
	{
		printf("Config: %i\n", config);
		for (power_of_two = power_of_two_start; power_of_two < power_of_two_number; power_of_two++)
		{
			printf("Size 2^%i MB\n", power_of_two);
			std::uint64_t inSize = std::uint64_t(int(pow(2, power_of_two))) * 1024 * 1024 / sizeof(float);
			for (int k = 0; k < small_test_number; k++)
			{
				float* inData;
				hipMallocManaged(&inData, inSize * sizeof(float));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <float> (std::rand()) / static_cast <float> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = Sort<float>::DataBlock;
				PassManager<float, Sort<float>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n");
		}
		for (multiplier_of_512 = multiplier_of_512_start; multiplier_of_512 < multiplier_of_512_number; multiplier_of_512++)
		{
			printf("Size 512*%i MB\n", multiplier_of_512);
			std::uint64_t inSize = std::uint64_t(512 * multiplier_of_512) * 1024 * 1024 / sizeof(float);
			limit = big_test_number;
			if (config == gpu) limit = small_test_number;
			for (int k = 0; k < limit; k++)
			{
				float* inData;
				hipMallocManaged(&inData, inSize * sizeof(float));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <float> (std::rand()) / static_cast <float> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = Sort<float>::DataBlock;
				PassManager<float, Sort<float>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n\n");
		}
	}

	printf("SORT - INT\n");
	for (config = config_start; config < config_number; config++)
	{
		printf("Config: %i\n", config);
		for (power_of_two = power_of_two_start; power_of_two < power_of_two_number; power_of_two++)
		{
			printf("Size 2^%i MB\n", power_of_two);
			std::uint64_t inSize = std::uint64_t(int(pow(2, power_of_two))) * 1024 * 1024 / sizeof(int);
			for (int k = 0; k < small_test_number; k++)
			{
				int* inData;
				hipMallocManaged(&inData, inSize * sizeof(int));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <int> (std::rand()) / static_cast <int> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = Sort<int>::DataBlock;
				PassManager<int, Sort<int>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n");
		}
		for (multiplier_of_512 = multiplier_of_512_start; multiplier_of_512 < multiplier_of_512_number; multiplier_of_512++)
		{
			printf("Size 512*%i MB\n", multiplier_of_512);
			std::uint64_t inSize = std::uint64_t(512 * multiplier_of_512) * 1024 * 1024 / sizeof(int);
			limit = big_test_number;
			if (config == gpu) limit = small_test_number;
			for (int k = 0; k < limit; k++)
			{
				int* inData;
				hipMallocManaged(&inData, inSize * sizeof(int));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <int> (std::rand()) / static_cast <int> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = Sort<int>::DataBlock;
				PassManager<int, Sort<int>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n\n");
		}
	}

	printf("REDUCE - FLOAT\n");
	for (config = config_start; config < config_number; config++)
	{
		printf("Config: %i\n", config);
		for (power_of_two = power_of_two_start; power_of_two < power_of_two_number; power_of_two++)
		{
			printf("Size 2^%i MB\n", power_of_two);
			std::uint64_t inSize = std::uint64_t(int(pow(2, power_of_two))) * 1024 * 1024 / sizeof(float);
			for (int k = 0; k < small_test_number; k++)
			{
				float* inData;
				hipMallocManaged(&inData, inSize * sizeof(float));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <float> (std::rand()) / static_cast <float> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = MaxReduction<float>::DataBlock;
				PassManager<float, MaxReduction<float>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n");
		}
		for (multiplier_of_512 = multiplier_of_512_start; multiplier_of_512 < multiplier_of_512_number; multiplier_of_512++)
		{
			printf("Size 512*%i MB\n", multiplier_of_512);
			std::uint64_t inSize = std::uint64_t(512 * multiplier_of_512) * 1024 * 1024 / sizeof(float);
			limit = big_test_number;
			if (config == gpu) limit = small_test_number;
			for (int k = 0; k < limit; k++)
			{
				float* inData;
				hipMallocManaged(&inData, inSize * sizeof(float));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <float> (std::rand()) / static_cast <float> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = MaxReduction<float>::DataBlock;
				PassManager<float, MaxReduction<float>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n\n");
		}
	}

	printf("REDUCE - INT\n");
	for (config = config_start; config < config_number; config++)
	{
		printf("Config: %i\n", config);
		for (power_of_two = power_of_two_start; power_of_two < power_of_two_number; power_of_two++)
		{
			printf("Size 2^%i MB\n", power_of_two);
			std::uint64_t inSize = std::uint64_t(int(pow(2, power_of_two))) * 1024 * 1024 / sizeof(int);
			for (int k = 0; k < small_test_number; k++)
			{
				int* inData;
				hipMallocManaged(&inData, inSize * sizeof(int));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <int> (std::rand()) / static_cast <int> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = MaxReduction<int>::DataBlock;
				PassManager<int, MaxReduction<int>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n");
		}
		for (multiplier_of_512 = multiplier_of_512_start; multiplier_of_512 < multiplier_of_512_number; multiplier_of_512++)
		{
			printf("Size 512*%i MB\n", multiplier_of_512);
			std::uint64_t inSize = std::uint64_t(512 * multiplier_of_512) * 1024 * 1024 / sizeof(int);
			limit = big_test_number;
			if (config == gpu) limit = small_test_number;
			for (int k = 0; k < limit; k++)
			{
				int* inData;
				hipMallocManaged(&inData, inSize * sizeof(int));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <int> (std::rand()) / static_cast <int> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = MaxReduction<int>::DataBlock;
				PassManager<int, MaxReduction<int>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n\n");
		}
	}

	printf("SEARCH - FLOAT\n");
	for (config = config_start; config < config_number; config++)
	{
		printf("Config: %i\n", config);
		for (power_of_two = power_of_two_start; power_of_two < power_of_two_number; power_of_two++)
		{
			printf("Size 2^%i MB\n", power_of_two);
			std::uint64_t inSize = std::uint64_t(int(pow(2, power_of_two))) * 1024 * 1024 / sizeof(float);
			for (int k = 0; k < small_test_number; k++)
			{
				float* inData;
				hipMallocManaged(&inData, inSize * sizeof(float));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <float> (std::rand()) / static_cast <float> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = BinarySearch<float>::DataBlock;
				PassManager<float, BinarySearch<float>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n");
		}
		for (multiplier_of_512 = multiplier_of_512_start; multiplier_of_512 < multiplier_of_512_number; multiplier_of_512++)
		{
			printf("Size 512*%i MB\n", multiplier_of_512);
			std::uint64_t inSize = std::uint64_t(512 * multiplier_of_512) * 1024 * 1024 / sizeof(float);
			limit = big_test_number;
			if (config == gpu) limit = small_test_number;
			for (int k = 0; k < limit; k++)
			{
				float* inData;
				hipMallocManaged(&inData, inSize * sizeof(float));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <float> (std::rand()) / static_cast <float> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = BinarySearch<float>::DataBlock;
				PassManager<float, BinarySearch<float>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n\n");
		}
	}

	printf("SEARCH - INT\n");
	for (config = config_start; config < config_number; config++)
	{
		printf("Config: %i\n", config);
		for (power_of_two = power_of_two_start; power_of_two < power_of_two_number; power_of_two++)
		{
			printf("Size 2^%i MB\n", power_of_two);
			std::uint64_t inSize = std::uint64_t(int(pow(2, power_of_two))) * 1024 * 1024 / sizeof(int);
			for (int k = 0; k < small_test_number; k++)
			{
				int* inData;
				hipMallocManaged(&inData, inSize * sizeof(int));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <int> (std::rand()) / static_cast <int> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = BinarySearch<int>::DataBlock;
				PassManager<int, BinarySearch<int>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n");
		}
		for (multiplier_of_512 = multiplier_of_512_start; multiplier_of_512 < multiplier_of_512_number; multiplier_of_512++)
		{
			printf("Size 512*%i MB\n", multiplier_of_512);
			std::uint64_t inSize = std::uint64_t(512 * multiplier_of_512) * 1024 * 1024 / sizeof(int);
			limit = big_test_number;
			if (config == gpu) limit = small_test_number;
			for (int k = 0; k < limit; k++)
			{
				int* inData;
				hipMallocManaged(&inData, inSize * sizeof(int));
				for (std::uint64_t i = 0; i < inSize; ++i)
				{
					inData[i] = static_cast <int> (std::rand()) / static_cast <int> (RAND_MAX);
				}

				auto start = std::chrono::steady_clock::now();
				using DataBlock = BinarySearch<int>::DataBlock;
				PassManager<int, BinarySearch<int>> pm(config);
				auto x = pm.run(DataBlock(inData, inSize));
				auto end = std::chrono::steady_clock::now();
				std::chrono::duration<double> elapsed_seconds = end - start;
				printf("%f\n", elapsed_seconds.count());
				hipFree(inData);
			}
			printf("\n\n");
		}
	}
	return 0;
}