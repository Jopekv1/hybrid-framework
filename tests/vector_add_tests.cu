#include "hip/hip_runtime.h"
#include "kernel.h"
#include "load_balancer.h"

#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>

constexpr uint64_t dataSize = 100000000;

void verify(int* dst, int size) {
	std::cout << "Veryfying data..." << std::endl;
	bool correct = true;
	int errCnt = 0;
	for (uint64_t i = 0; i < size; i++) {
		if (dst[i] != 5764801) {
			correct = false;
			errCnt++;
			//std::cout << i << std::endl;
		}
	}
	if (correct) {
		std::cout << "Results correct" << std::endl;
	}
	else {
		std::cout << "!!!!! ERROR !!!!!" << std::endl;
	}
}

__global__
void add(int n, int* src, int* dst) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n) {
		dst[index] = (int)pow((double)src[index], (double)dst[index]);
	}
}

class VecAddKernel : public Kernel {
public:

	VecAddKernel() {
		std::cout << "Initializing data..." << std::endl;

		hipHostMalloc(&srcHost, dataSize * sizeof(int));
		hipHostMalloc(&dstHost, dataSize * sizeof(int));

		hipMalloc(&src, dataSize * sizeof(int));
		hipMalloc(&dst, dataSize * sizeof(int));

		for (uint64_t i = 0; i < dataSize; i++) {
			srcHost[i] = 7;
			dstHost[i] = 8;
		}

		hipStreamCreate(&ownStream);

		hipMemcpyAsync(src, srcHost, dataSize * sizeof(int), hipMemcpyHostToDevice, ownStream);
		hipMemcpyAsync(dst, dstHost, dataSize * sizeof(int), hipMemcpyHostToDevice, ownStream);

		std::cout << "Data initialized" << std::endl;
	}

	~VecAddKernel() {
		hipFree(dst);
		hipFree(src);

		hipHostFree(dstHost);
		hipHostFree(srcHost);

		hipStreamDestroy(ownStream);
	}

	void runCpu(int workItemId, int workGroupSize) override {
		for (int i = workItemId; i < workItemId + workGroupSize; i++) {
			dstHost[i] = (int)pow((double)srcHost[i], (double)dstHost[i]);
		}
	};

	void runGpu(int deviceId, int workItemId, int workGroupSize) override {
		int blockSize = 1024;
		int numBlocks = (workGroupSize + blockSize - 1) / blockSize;
		add<<<numBlocks, blockSize, 0, ownStream>>>(workGroupSize, src + workItemId, dst + workItemId);
		hipMemcpyAsync(dstHost + workItemId, dst + workItemId, workGroupSize * sizeof(int), hipMemcpyDeviceToHost, ownStream);
	};

	int* src = nullptr;
	int* dst = nullptr;
	int* srcHost = nullptr;
	int* dstHost = nullptr;

	hipStream_t ownStream;
};

class VectorAddFixture : public ::testing::TestWithParam<std::tuple<uint64_t, uint64_t, int>> {
public:

	void SetUp() override {
		std::tie(workGroupSize, gpuWorkGroups, numThreads) = GetParam();

		std::cout << "Test params: workGroupSize: " << workGroupSize << ", gpuWorkGroups: " << gpuWorkGroups << ", numThread: " << numThreads << std::endl;

		if (gpuWorkGroups * workGroupSize >= dataSize) {
			std::cout << "!!!!!!!!!!!!!!!!! GPU COVERS WHOLE DATA !!!!!!!!!!!!!!!!!!!!!!" << std::endl;
		}
	}

	uint64_t workGroupSize = 0;
	uint64_t gpuWorkGroups = 0;
	int numThreads = 0;
};

TEST_P(VectorAddFixture, hybrid) {
	VecAddKernel kernel;

	LoadBalancer balancer(workGroupSize, gpuWorkGroups, numThreads);

	auto start = std::chrono::steady_clock::now();
	balancer.execute(&kernel, dataSize);
	auto end = std::chrono::steady_clock::now();

	std::chrono::duration<double> elapsed_seconds = end - start;
	std::cout << "Hybrid time: " << elapsed_seconds.count() << "s\n";

	verify(kernel.dstHost, dataSize);
}

static uint64_t workGroupSizesValues[] = {
	10,
	100,
	1000,
	10000,
	100000 };

static uint64_t gpuWorkGroupsValues[] = {
	100,
	1000,
	10000,
	20000,
	50000,
	100000 };

static int numThreadsValues[] = {
	2,
	4,
	6,
	8 };

INSTANTIATE_TEST_SUITE_P(VectorAdd,
	VectorAddFixture,
	::testing::Combine(
		::testing::ValuesIn(workGroupSizesValues),
		::testing::ValuesIn(gpuWorkGroupsValues),
		::testing::ValuesIn(numThreadsValues)));

TEST(VectorAdd, gpu) {
	std::cout << "Initializing data..." << std::endl;

	int* src = nullptr;
	int* dst = nullptr;
	int* srcHost = nullptr;
	int* dstHost = nullptr;

	hipStream_t ownStream;

	hipHostMalloc(&srcHost, dataSize * sizeof(int));
	hipHostMalloc(&dstHost, dataSize * sizeof(int));

	hipMalloc(&src, dataSize * sizeof(int));
	hipMalloc(&dst, dataSize * sizeof(int));

	for (uint64_t i = 0; i < dataSize; i++) {
		srcHost[i] = 7;
		dstHost[i] = 8;
	}

	hipStreamCreate(&ownStream);

	hipMemcpyAsync(src, srcHost, dataSize * sizeof(int), hipMemcpyHostToDevice, ownStream);
	hipMemcpyAsync(dst, dstHost, dataSize * sizeof(int), hipMemcpyHostToDevice, ownStream);

	std::cout << "Data initialized" << std::endl;

	int blockSize = 1024;
	int numBlocks = (dataSize + blockSize - 1) / blockSize;

	auto start = std::chrono::steady_clock::now();
	add << <numBlocks, blockSize, 0, ownStream >> > (dataSize, src, dst);
	hipMemcpyAsync(dstHost, dst, dataSize * sizeof(int), hipMemcpyDeviceToHost, ownStream);
	hipDeviceSynchronize();
	auto end = std::chrono::steady_clock::now();

	std::chrono::duration<double> elapsed_seconds = end - start;
	std::cout << "GPU time: " << elapsed_seconds.count() << "s\n";

	verify(dstHost, dataSize);

	hipFree(dst);
	hipFree(src);

	hipHostFree(dstHost);
	hipHostFree(srcHost);

	hipStreamDestroy(ownStream);
}