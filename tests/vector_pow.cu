#include "hip/hip_runtime.h"
#include "kernel.h"
#include "load_balancer.h"

#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <random>
#include <cmath>

constexpr uint64_t dataSize = 2684353186 / 4;
constexpr uint64_t gpuAllocSize = 1073741824 / 4;

const double e = std::exp(1.0);

void verifyVectorPow(double* dst, double* src) {
	std::cout << "Veryfying data..." << std::endl;
	bool correct = true;
	for (uint64_t i = 0; i < dataSize; i++) {
		auto expercted = pow(src[i], e);
		auto value = dst[i];
		if (value < expercted - 0.01 || value > expercted + 0.01) {
			correct = false;
		}
	}
	if (correct) {
		std::cout << "Results correct" << std::endl;
	}
	else {
		std::cout << "!!!!! ERROR !!!!!" << std::endl;
		throw std::exception();
	}
}

__global__
void pow(int n, double* src, double* dst, double e) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n) {
		dst[index] = pow(src[index], e);
	}
}

class VecPowKernel : public Kernel {
public:

	VecPowKernel() {
		std::cout << "Initializing data..." << std::endl;

		hipHostMalloc(&srcHost, dataSize * sizeof(double));
		hipHostMalloc(&dstHost, dataSize * sizeof(double));

		hipMalloc(&src, gpuAllocSize * sizeof(double));
		hipMalloc(&dst, gpuAllocSize * sizeof(double));

		double lower_bound = 0;
		double upper_bound = 100000;
		std::uniform_real_distribution<double> distribution(lower_bound, upper_bound);
		std::default_random_engine randomEngine;

		for (uint64_t i = 0; i < dataSize; i++) {
			srcHost[i] = distribution(randomEngine);
			dstHost[i] = 0;
		}

		hipStreamCreate(&ownStream);

		std::cout << "Data initialized" << std::endl;
	}

	~VecPowKernel() {
		hipFree(dst);
		hipFree(src);

		hipHostFree(dstHost);
		hipHostFree(srcHost);

		hipStreamDestroy(ownStream);
	}

	void runCpu(uint64_t workItemId, uint64_t workGroupSize) override {
		for (int i = workItemId; i < workItemId + workGroupSize; i++) {
			dstHost[i] = pow(srcHost[i], e);
		}
	};

	void runGpu(uint64_t deviceId, uint64_t workItemId, uint64_t workGroupSize) override {
		uint64_t i = 0;
		while (i < workGroupSize) {
			auto size = gpuAllocSize;
			if (i + gpuAllocSize > workGroupSize) {
				size = workGroupSize - i;
			}

			int blockSize = 1024;
			int numBlocks = (size + blockSize - 1) / blockSize;

			hipMemcpyAsync(src, srcHost + workItemId + i, size * sizeof(double), hipMemcpyHostToDevice, ownStream);
			pow<<<numBlocks, blockSize, 0, ownStream>>>(size, src, dst, e);
			hipMemcpyAsync(dstHost + workItemId + i, dst, size * sizeof(double), hipMemcpyDeviceToHost, ownStream);
			hipStreamSynchronize(ownStream);

			i += size;
		}
	};

	double* src = nullptr;
	double* dst = nullptr;
	double* srcHost = nullptr;
	double* dstHost = nullptr;

	hipStream_t ownStream;
};

class VectorPowFixture : public ::testing::TestWithParam<std::tuple<uint64_t, uint64_t, int>> {
public:

	void SetUp() override {
		std::tie(workGroupSize, gpuWorkGroups, numThreads) = GetParam();

		std::cout << "Test params: workGroupSize: " << workGroupSize << ", gpuWorkGroups: " << gpuWorkGroups << ", numThread: " << numThreads << std::endl;

		if (gpuWorkGroups * workGroupSize >= dataSize) {
			std::cout << "!!!!!!!!!!!!!!!!! GPU COVERS WHOLE DATA !!!!!!!!!!!!!!!!!!!!!!" << std::endl;
			//GTEST_SKIP();
		}
		if (gpuWorkGroups * workGroupSize >= gpuAllocSize) {
			std::cout << "!!!!!!!!!!!!!!!!! GPU PACKAGE BIGGER THAN GPU ALLOC SIZE !!!!!!!!!!!!!!!!!!!!!!" << std::endl;
			//GTEST_SKIP();
		}
	}

	uint64_t workGroupSize = 0;
	uint64_t gpuWorkGroups = 0;
	int numThreads = 0;
};

TEST_P(VectorPowFixture, hybrid) {
	VecPowKernel kernel;

	LoadBalancer balancer(workGroupSize, gpuWorkGroups, numThreads);

	auto start = std::chrono::steady_clock::now();
	balancer.execute(&kernel, dataSize);
	auto end = std::chrono::steady_clock::now();

	std::chrono::duration<double> elapsed_seconds = end - start;
	std::cout << "Hybrid time: " << elapsed_seconds.count() << "s\n";

	//verifyVectorPow(kernel.dstHost, kernel.srcHost);

	auto hybridFile = fopen("results_hybrid.txt", "a");
	fprintf(hybridFile, "VectorPow %llu %llu %d %Lf\n", workGroupSize, gpuWorkGroups, numThreads, elapsed_seconds.count());
	fclose(hybridFile);
}

static uint64_t workGroupSizesValues[] = {
	10,
	100,
	1000,
	10000,
	100000 };

static uint64_t gpuWorkGroupsValues[] = {
	100,
	1000,
	10000,
	20000,
	50000,
	100000 };

static int numThreadsValues[] = {
	2,
	4,
	6,
	8 };

INSTANTIATE_TEST_SUITE_P(VectorPow,
	VectorPowFixture,
	::testing::Combine(
		::testing::ValuesIn(workGroupSizesValues),
		::testing::ValuesIn(gpuWorkGroupsValues),
		::testing::ValuesIn(numThreadsValues)));

TEST(VectorPow, gpu) {
	VecPowKernel kernel;

	auto start = std::chrono::steady_clock::now();

	kernel.runGpu(0u, 0u, dataSize);

	auto end = std::chrono::steady_clock::now();

	std::chrono::duration<double> elapsed_seconds = end - start;
	std::cout << "GPU time: " << elapsed_seconds.count() << "s\n";

	//verifyVectorPow(kernel.dstHost, kernel.srcHost);

	auto gpuFile = fopen("results_gpu.txt", "a");
	fprintf(gpuFile, "VectorPow %Lf\n", elapsed_seconds.count());
	fclose(gpuFile);
}